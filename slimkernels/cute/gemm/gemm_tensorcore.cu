#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <cute/tensor.hpp>
#include <cute/underscore.hpp>
#include <cute/numeric/integral_constant.hpp>
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/kernel/gemm.h"
#include "gemm.h"

using namespace cute;

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = float;                   // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = float;          // <- data type of elements in input matrix A
using ElementInputB = float;          // <- data type of elements in input matrix B
using ElementOutput = float;                        // <- data type of elements in output matrix D

const int  THREADS_PER_WARP=32;

struct GemmConfig {
    static constexpr int BLOCK_DIM_M = 128;
    static constexpr int BLOCK_DIM_N = 128;
    static constexpr int BLOCK_DIM_K = 16;
    static constexpr int WARP_TILE_DIM_M = 64;
    static constexpr int WARP_TILE_DIM_N = 64;
    static constexpr int THREAD_NUM = 256;
    static constexpr int TENSOR_CORE_M = 16;
    static constexpr int TENSOR_CORE_N = 8;
    static constexpr int TENSOR_CORE_K = 16;
};

template <typename GemmConfig>
struct CuTeGemmConfig {
  // tile configuration
  static constexpr int kTileM = GemmConfig::BLOCK_DIM_M;
  static constexpr int kTileN = GemmConfig::BLOCK_DIM_N;
  static constexpr int kTileK = GemmConfig::BLOCK_DIM_K;
  static constexpr int kStage = 5;

  static constexpr int kShmLoadSwizzleM = 3;
  static constexpr int kShmLoadSwizzleS = 3;
  static constexpr int kShmLoadSwizzleB = 3;

  // define shared memory layout
  using SmemLayoutAtom = decltype(composition(
      Swizzle<kShmLoadSwizzleB, kShmLoadSwizzleM, kShmLoadSwizzleS>{},
      make_layout(make_shape(Int<8>{}, Int<kTileK>{}),
                  make_stride(Int<kTileK>{}, Int<1>{}))));
  using SmemLayoutA = decltype(
      tile_to_shape(SmemLayoutAtom{},
                    make_shape(Int<kTileM>{}, Int<kTileK>{}, Int<kStage>{})));
  using SmemLayoutB = decltype(
      tile_to_shape(SmemLayoutAtom{},
                    make_shape(Int<kTileN>{}, Int<kTileK>{}, Int<kStage>{})));

  // define a mma tile 32X32
  using mma_op = SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;
  static constexpr int kMmaEURepeatM = 2;
  static constexpr int kMmaEURepeatN = 2;
  static constexpr int kMmaEURepeatK = 1;
  using mma_atom_shape = mma_traits::Shape_MNK;
  static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
  static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
  static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
  using MMA_EU_RepeatT = decltype(make_layout(make_shape(
      Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
  using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
  using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{})); // 32X32 tile with 8 16X8X16 tensorcores

  // define copy from global to shared memory
  using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
  using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
  using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
  using G2SCopyA =
      decltype(make_tiled_copy(g2s_copy_atom{},
                               make_layout(make_shape(Int<32>{}, Int<4>{}),
                                           make_stride(Int<4>{}, Int<1>{})),
                               make_layout(make_shape(Int<1>{}, Int<8>{}))));
  using G2SCopyB = G2SCopyA;

  // define copy from shared memory to registers
  using s2r_copy_op = SM75_U32x4_LDSM_N;
  using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
  using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
  using S2RCopyAtomA = s2r_copy_atom;
  using S2RCopyAtomB = s2r_copy_atom;

  // epilogue: register to global via shared memory
  using SmemLayoutAtomC = decltype(composition(
      Swizzle<2, 3, 3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                      make_stride(Int<kMmaPN>{}, Int<1>{}))));
  using SmemLayoutC = decltype(tile_to_shape(
      SmemLayoutAtomC{},
      make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

  static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                    size(SmemLayoutC{}),
                "C shared memory request is large than A's one pipe");

  using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;

  using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
  using S2GCopyC =
      decltype(make_tiled_copy(S2GCopyAtomC{},
                               make_layout(make_shape(Int<32>{}, Int<4>{}),
                                           make_stride(Int<4>{}, Int<1>{})),
                               make_layout(make_shape(Int<1>{}, Int<8>{}))));

  static constexpr int kThreadNum = size(MMA{});
  static constexpr int shm_size_AB =
      cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
  static constexpr int shm_size_C = cute::cosize(SmemLayoutC{});

  static constexpr int kShmSize =
      cute::max(shm_size_AB, shm_size_C) * sizeof(T);
};


template<typename Config>
__global__ void GEMM_MMA(MMAarguments arg){
    auto shape_MNK = make_shape(arg.problem_size.m(), arg.problem_size.n(), arg.problem_size.k());
    using namespace cute;
    using X = Underscore;

    using T = typename Config::T;
    using SmemLayoutA = typename Config::SmemLayoutA;
    using SmemLayoutB = typename Config::SmemLayoutB;
    using SmemLayoutC = typename Config::SmemLayoutC;
    using TiledMMA = typename Config::MMA;

    using S2RCopyAtomA = typename Config::S2RCopyAtomA;
    using S2RCopyAtomB = typename Config::S2RCopyAtomB;
    using G2SCopyA = typename Config::G2SCopyA;
    using G2SCopyB = typename Config::G2SCopyB;
    using R2SCopyAtomC = typename Config::R2SCopyAtomC;
    using S2GCopyAtomC = typename Config::S2GCopyAtomC;
    using S2GCopyC = typename Config::S2GCopyC;

    constexpr int kTileM = Config::kTileM;
    constexpr int kTileN = Config::kTileN;
    constexpr int kTileK = Config::kTileK;
    constexpr int kStage = Config::kStage;
    
    constexpr int kTileM = Config::kTileM;
  constexpr int kTileN = Config::kTileN;
  constexpr int kTileK = Config::kTileK;
  constexpr int kStage = Config::kStage;

  extern __shared__ T shm_data[];

  T *Ashm = shm_data;
  T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

  int idx = threadIdx.x;
  int ix = blockIdx.x;
  int iy = blockIdx.y;

  // use Tensor notation to represent device pointer + dimension
  Tensor A = make_tensor(make_gmem_ptr((T *)Aptr), make_shape(m, k),
                         make_stride(k, Int<1>{}));  // (M, K)
  Tensor B = make_tensor(make_gmem_ptr((T *)Bptr), make_shape(n, k),
                         make_stride(k, Int<1>{}));  // (N, K)
  Tensor D = make_tensor(make_gmem_ptr((T *)Dptr), make_shape(m, n),
                         make_stride(n, Int<1>{}));  // (M, N)

  // slice the tensor to small one which is used for current thread block.
  Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}),
                         make_coord(iy, _));  // (kTileM, kTileK, k)
  Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}),
                         make_coord(ix, _));  // (kTileN, kTileK, k)
  Tensor gD = local_tile(D, make_tile(Int<kTileM>{}, Int<kTileN>{}),
                         make_coord(iy, ix));  // (kTileM, kTileN)

  // shared memory
  auto sA = make_tensor(make_smem_ptr(Ashm),
                        SmemLayoutA{});  // (kTileM, kTileK, kStage)
  auto sB = make_tensor(make_smem_ptr(Bshm),
                        SmemLayoutB{});  // (kTileN, kTileK, kStage)

  // dispatch TileA/TileB/TileC mma tensor into thread fragment via partition
  // method
  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(idx);
  auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
  auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
  auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)

  // fill zero for accumulator
  clear(tCrD);

  // gmem -cp.async-> shm -ldmatrix-> reg
  auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
  auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
  auto tAsA = s2r_thr_copy_a.partition_S(sA);  // ? (CPY, CPY_M, CPY_K, kStage)
  auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA);  // ? (CPY, CPY_M, CPY_K)

  auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
  auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
  auto tBsB = s2r_thr_copy_b.partition_S(sB);  // ? (CPY, CPY_M, CPY_K, kStage)
  auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB);  // ? (CPY, CPY_M, CPY_K)

  G2SCopyA g2s_tiled_copy_a;
  auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
  auto tAgA_copy = g2s_thr_copy_a.partition_S(gA);  // (CPY, CPY_M, CPY_K, k)
  auto tAsA_copy =
      g2s_thr_copy_a.partition_D(sA);  // (CPY, CPY_M, CPY_K, kStage)

  G2SCopyB g2s_tiled_copy_b;
  auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
  auto tBgB_copy = g2s_thr_copy_b.partition_S(gB);  // (CPY, CPY_N, CPY_K, k)
  auto tBsB_copy =
      g2s_thr_copy_b.partition_D(sB);  // (CPY, CPY_N, CPY_K, kStage)

  int itile_to_read = 0;
  int ismem_read = 0;
  int ismem_write = 0;

  // submit kStage - 1 tile
  // gmem -> shm
#pragma unroll
  for (int istage = 0; istage < kStage - 1; ++istage) {
    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, istage),
               tAsA_copy(_, _, _, istage));
    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, istage),
               tBsB_copy(_, _, _, istage));
    cp_async_fence();

    ++itile_to_read;
    ++ismem_write;
  }

  // wait one submitted gmem->smem done
  cp_async_wait<kStage - 2>();
  __syncthreads();

  int ik = 0;
  // smem -> reg
  cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik, ismem_read), tCrA_view(_, _, ik));
  cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik, ismem_read), tCrB_view(_, _, ik));

  // loop over k: i. load tile, ii. mma
  int ntile = k / kTileK;
#pragma unroll 1
  for (int itile = 0; itile < ntile; ++itile) {
    int nk = size<2>(tCrA);

#pragma unroll
    for (int ik = 0; ik < nk; ++ik) {
      int ik_next = (ik + 1) % nk;

      if (ik == nk - 1) {
        cp_async_wait<kStage - 2>();
        __syncthreads();

        ismem_read = (ismem_read + 1) % kStage;
      }

      // shm -> reg s[itile][ik + 1] -> r[ik + 1]
      cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik_next, ismem_read),
                 tCrA_view(_, _, ik_next));
      cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik_next, ismem_read),
                 tCrB_view(_, _, ik_next));

      if (ik == 0) {
        if (itile_to_read < ntile) {
          cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile_to_read),
                     tAsA_copy(_, _, _, ismem_write));
          cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile_to_read),
                     tBsB_copy(_, _, _, ismem_write));

          ++itile_to_read;
          ismem_write = (ismem_write + 1) % kStage;
        }

        cp_async_fence();
      }

      cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
    }  // for ik
  }    // itile

  // use less shared memory as a scratchpad tile to use large wide instuction
  // Dreg -> shm -> reg -> global
  auto sC = make_tensor(sA(_, _, ismem_read).data(), SmemLayoutC{});

  auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
  auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
  auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);   // (CPY, CPY_M, CPY_N)
  auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC);  // (CPY, _1, _1, pipe)

  S2GCopyC s2g_tiled_copy_c;
  auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
  auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC);  // (CPY, _1, _1, pipe)
  auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD);  // (CPY, CPY_M, CPY_N)

  auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g);  // (CPY_, CPY_MN)
  auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s);  // (CPY_, CPY_MN)

  int step = size<3>(tCsC_r2s);  // pipe
#pragma unroll
  for (int i = 0; i < size<1>(tCrC_r2sx); i += step) {
    // reg -> shm
#pragma unroll
    for (int j = 0; j < step; ++j) {
      // we add a temp tensor to cope with accumulator and output data type
      // difference
      auto t = make_tensor_like<T>(tCrC_r2sx(_, i + j));
      cute::copy(tCrC_r2sx(_, i + j), t);

      cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
    }
    __syncthreads();

#pragma unroll
    // shm -> global
    for (int j = 0; j < step; ++j) {
      cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i + j));
    }

    __syncthreads();
  }

}

void launch_GEMM_MMA(MMAarguments &arg){
    dim3 grid,block;
    grid.x = (arg.problem_size.n()+GemmConfig::BLOCK_DIM_N-1)/GemmConfig::BLOCK_DIM_N;
    grid.y = (arg.problem_size.m()+GemmConfig::BLOCK_DIM_M-1)/GemmConfig::BLOCK_DIM_M;
    grid.z = 1;

    block.x = CuTeGemmConfig<GemmConfig>::kThreadNum;
    block.y = 1;
    block.z = 1;

    GEMM_MMA<CuTeGemmConfig<GemmConfig>><<<grid,block>>>(arg);
    
    
}